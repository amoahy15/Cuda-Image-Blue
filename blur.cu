#include "hip/hip_runtime.h"
/*By eliminating redundant calculations and simplifying the memory access pattern, the blur program's performance can be enhanced. To reduce global memory accesses and make input data available to numerous threads, we specifically used tiling. The input image is divided into tiles so that each thread can focus on a different area of the image while reusing information from surrounding threads. As a result, the number of global memory accesses is decreased, and the memory access pattern is enhanced. We can see this in the run times of both programs, the program using tiling and shared memory was significantly faster than the naive method.
 */

#include <iostream>
#include <vector>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>

#define BLUR_SIZE 16 // size of surrounding image is 2X this

#include "bitmap_image.hpp"

using namespace std;

__global__ void blurKernel(uchar3 *in, uchar3 *out, int width, int height)
{

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < width && row < height)
    {
        int3 pixVal;
        pixVal.x = 0;
        pixVal.y = 0;
        pixVal.z = 0;
        int pixels = 0;

        // get the average of the surrounding 2xBLUR_SIZE x 2xBLUR_SIZE box
        for (int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE + 1; blurRow++)
        {
            for (int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE + 1; blurCol++)
            {

                int curRow = row + blurRow;
                int curCol = col + blurCol;

                // verify that we have a valid image pixel
                if (curRow > -1 && curRow < height && curCol > -1 && curCol < width)
                {
                    pixVal.x += in[curRow * width + curCol].x;
                    pixVal.y += in[curRow * width + curCol].y;
                    pixVal.z += in[curRow * width + curCol].z;
                    pixels++; // keep track of number of pixels in the accumulated total
                }
            }
        }

        // write our new pixel value out
        out[row * width + col].x = (unsigned char)(pixVal.x / pixels);
        out[row * width + col].y = (unsigned char)(pixVal.y / pixels);
        out[row * width + col].z = (unsigned char)(pixVal.z / pixels);
    }
}
int main(int argc, char **argv)
{
    if (argc != 2)
    {
        cerr << "format: " << argv[0] << " { 24-bit BMP Image Filename }" << endl;
        exit(1);
    }
    string input_filename(argv[1]);
    string output_filename = "./" + input_filename.substr(0, input_filename.find_last_of(".")) + "_blurred.bmp";


    bitmap_image bmp(argv[1]);

    if (!bmp)
    {
        cerr << "Image not found" << endl;
        exit(1);
    }

    int height = bmp.height();
    int width = bmp.width();
    int image_size = height * width;

    cout << "Image dimensions:" << endl;
    cout << "height: " << height << " width: " << width << endl;

    cout << "Converting " << argv[1] << " from color to grayscale..." << endl;

    // Transform image into vector of doubles
    vector<uchar3> input_image;
    rgb_t color;
    for (int x = 0; x < width; x++)
    {
        for (int y = 0; y < height; y++)
        {
            bmp.get_pixel(x, y, color);
            input_image.push_back({color.red, color.green, color.blue});
        }
    }

    vector<uchar3> output_image(input_image.size());

    uchar3 *d_in, *d_out;
    int img_size = (input_image.size() * sizeof(char) * 3);
    hipMalloc(&d_in, img_size);
    hipMalloc(&d_out, img_size);

    hipMemcpy(d_in, input_image.data(), img_size, hipMemcpyHostToDevice);
    hipMemcpy(d_out, input_image.data(), img_size, hipMemcpyHostToDevice);
    hipEvent_t start;
    hipEventCreate(&start);
    hipEvent_t stop;
    hipEventCreate(&stop);
    // start timer
    hipEventRecord(start, 0);

    // Switched the height and width in the block and kernel call
    //  TODO: Fill in the correnct blockSize and gridSize
    dim3 dimGrid(ceil(width / 16.0) + 1, ceil(height / 16.0) + 1, 1);
    dim3 dimBlock(16, 16, 1);

    blurKernel<<<dimGrid, dimBlock>>>(d_in, d_out, height, width);
    hipDeviceSynchronize();

    hipMemcpy(output_image.data(), d_out, img_size, hipMemcpyDeviceToHost);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float diff;
    hipEventElapsedTime(&diff, start, stop);
    printf("time: %f ms\n", diff);
   std::ofstream output_file("./time.csv", std::ios::out | std::ios::app);
output_file << diff << "," << image_size << std::endl;
    output_file.close();

    // deallocate timers
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Set updated pixels

    for (int x = 0; x < width; x++)
    {
        for (int y = 0; y < height; y++)
        {
            int pos = x * height + y;
            bmp.set_pixel(x, y, output_image[pos].x, output_image[pos].y, output_image[pos].z);
        }
    }

    cout << "Conversion complete." << endl;

    bmp.save_image(output_filename);

    hipFree(d_in);
    hipFree(d_out);
}
